#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <math.h>
#include <malloc.h>
#include <stdlib.h>

#define PRECISION float

#define  max 999 
#define  maa   9 

/*Block size depends on maximum threads*/
#define BLOCK_SIZE_X  16
#define BLOCK_SIZE_Y  16

/* Parameters */
int imax , jmax ;
int imax2 , jmax2 ;
int ima  , jma  ;
int isd  , jsd  ;
int ied  , jed  ;
int numGPUs ;

long ncye , nwri ;

PRECISION reyn , rtau , csou , rcsu ;
PRECISION pini , uini , vini , uwui , runi ;

PRECISION ex[maa] , ey[maa] , we[maa] ;
PRECISION fn[max][max][maa] , fe[max][max][maa] , fp[max][max][maa] ;
PRECISION pn[max][max] , un[max][max] , vn[max][max] ;
PRECISION xg[max][max] , yg[max][max] ;


#define PTR(i, j) (imax2 * j + i)
#define PTRQ(i, j, k) ((j*imax2*maa)+(i*maa)+k) 


__constant__ PRECISION dex[maa];
__constant__ PRECISION dey[maa];
__constant__ PRECISION dwe[maa];

struct Conditions {
	int imax, jmax ;
	int imax2, jmax2 ;
	int isd, ied, jsd, jed ;
	PRECISION rtau, rcsu, runi, uwui ;
};

//ŠÖ”‚Ìƒvƒƒgƒ^ƒCƒvéŒ¾//
void iniset(void) ;
void inicon(void) ;
void solver(void) ;


//********************************‘æˆêˆ—‹æ‰æ**********************************//
//
//
//******************************************************************************//
/*__global__ void matrix1(PRECISION *dfe, PRECISION* dfp, PRECISION* dfn,
			            struct Conditions *d_cond) */ // original definition

__global__ void matrix1(PRECISION *dfe, PRECISION* dfp, PRECISION* dfn, PRECISION *dmarker,
						PRECISION *dun, PRECISION *dvn, PRECISION *dpn,
			            struct Conditions *d_cond)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x  ;
	const int j = blockIdx.y * blockDim.y + threadIdx.y  ;

	//ƒOƒ[ƒoƒ‹•Ï”‚ÍGPU“à‚ÉŠ±Â‚Å‚«‚È‚¢//
	const int imax2 = d_cond->imax2 ;
	const PRECISION rtau = d_cond->rtau  ;
	
	const PRECISION runi = d_cond->runi;

	const int imax = d_cond->imax ;
	const int jmax = d_cond->jmax ;

	if(i==0){
	}
	else if(i==imax){
	}
	else if(j==0){
	}
	else if(j==jmax){
	}
	else{
		for(int k=0;k<maa;k++){
			int ii = i - int(dex[k]) ;
			int jj = j - int(dey[k]) ;
			const int pointer = PTRQ(ii,jj,k) ;
			const int pointer2 = PTR(ii,jj);
//add external forcing here
			/*	euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
		qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
		dfe[PTRQ(i, j, k)] = dwe[k] * (dpn[PTR(i, j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
		dfn[PTRQ(i, j, k)] = dfp[PTRQ(i, j, k)] ; */


//dfp[PTRQ(i, j, k)] = dfn[pointer]+ rtau * (dfe[pointer] - dfn[pointer]) + dpn[pointer]*dmarker[pointer]*(dwe[k]*( -dex[k]*dun[pointer] -dey[k]*dvn[pointer] )) ;
dfp[PTRQ(i, j, k)] = dfn[pointer]+ rtau * (dfe[pointer] - dfn[pointer]) + runi*dmarker[pointer2]*(dwe[k]*( -dex[k]*dun[pointer2] -dey[k]*dvn[pointer2] )) ;

/* original line */
//dfp[PTRQ(i, j, k)] = dfn[pointer]+ rtau * (dfe[pointer] - dfn[pointer]);
		}
	}
	__syncthreads() ;
}


//********************************‘æ“ñˆ—‹æ‰æ**********************************//
//
//
//******************************************************************************//
__global__ void matrix2(PRECISION *dfp, 
						PRECISION *dpn, PRECISION *dun, PRECISION *dvn,
						PRECISION *ddps, PRECISION *dduv,
						struct Conditions *d_cond)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x  ;
	const int j = blockIdx.y * blockDim.y + threadIdx.y  ;

	const int imax2 = d_cond->imax2 ;
	const PRECISION rcsu  = d_cond->rcsu  ;

	const int imax = d_cond->imax ;
	const int jmax = d_cond->jmax ;

	if(i==0){
	}
	else if(i==imax){
	}
	else if(j==0){
	}
	else if(j==jmax){
	}
	else{
		PRECISION pss = 0.0 ;
		PRECISION uss = 0.0 ; 
		PRECISION vss = 0.0 ;

		for(int k=0;k<maa;k++){
			pss +=          dfp[PTRQ(i, j, k)] ;
			uss += dex[k] * dfp[PTRQ(i, j, k)] ;
			vss += dey[k] * dfp[PTRQ(i, j, k)] ;
		}
		pss =        pss ;
		uss = rcsu * uss ;
		vss = rcsu * vss ;
		
		ddps[PTR(i, j)] = (pss - dpn[PTR(i, j)]) * (pss - dpn[PTR(i, j)]) ;
		dduv[PTR(i, j)] = (uss - dun[PTR(i, j)]) * (uss - dun[PTR(i, j)]) 
						+ (vss - dvn[PTR(i, j)]) * (vss - dvn[PTR(i, j)]) ;
		dpn[PTR(i, j)] = pss ;
		dun[PTR(i, j)] = uss ;
		dvn[PTR(i, j)] = vss ;
	}
	__syncthreads() ;
}


//********************************‘æŽOˆ—‹æ‰æ**********************************//
//
//
//******************************************************************************//
__global__ void matrix3(PRECISION *dfe, PRECISION *dfp, PRECISION *dfn,
						PRECISION *dpn, PRECISION *dun, PRECISION *dvn,
						struct Conditions *d_cond)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x  ;
	const int j = blockIdx.y * blockDim.y + threadIdx.y  ;

	const int imax2 = d_cond->imax2 ;
	const PRECISION runi = d_cond->runi ;

	PRECISION euv, qau ;
	
	for(int k=0;k<maa;k++){
		euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
		qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
		dfe[PTRQ(i, j, k)] = dwe[k] * (dpn[PTR(i, j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
		dfn[PTRQ(i, j, k)] = dfp[PTRQ(i, j, k)] ;
	}
	__syncthreads() ;
}


//******************************‘æˆê‹«ŠEˆ—‹æ‰æ********************************//
//
//
//******************************************************************************//
__global__ void CUDAboundp(PRECISION *dpn, PRECISION *dun, PRECISION *dvn,
						   struct Conditions *d_cond)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x  ;
	const int j = blockIdx.y * blockDim.y + threadIdx.y  ;

	const int isd   = d_cond->isd ;
	const int ied   = d_cond->ied ;
	const int jsd   = d_cond->jsd ;
	const int jed   = d_cond->jed ;
	const int imax  = d_cond->imax ;
	const int jmax  = d_cond->jmax ;
	const int imax2 = d_cond->imax2 ;
	const PRECISION uwui = d_cond->uwui ;

	if(i == 0 && j == 0){//¶‰º//
		int i1 = i + 1 ; int j1 = j + 1 ;
		int i2 = i + 2 ; int j2 = j + 2 ;
		dpn[PTR(i, j)] = (9.0 * dpn[PTR(i1,j1)] - dpn[PTR(i2,j2)]) / 8.0 ;
		dun[PTR(i, j)] = 0.0 ;
		dvn[PTR(i, j)] = 0.0 ;
	}
	
	else if(i == imax && j == 0){//‰E‰º//
		int i1 = i - 1 ; int j1 = j + 1 ;
		int i2 = i - 2 ; int j2 = j + 2 ;
		dpn[PTR(i, j)] = (9.0 * dpn[PTR(i1,j1)] - dpn[PTR(i2,j2)]) / 8.0 ;
		dun[PTR(i, j)] = 0.0 ;
		dvn[PTR(i, j)] = 0.0 ;
	}
	
	else if(i == 0 && j == jmax){//¶ã//
		int i1 = i + 1 ; int j1 = j - 1 ;
		int i2 = i + 2 ; int j2 = j - 2 ;
		dpn[PTR(i, j)] = (9.0 * dpn[PTR(i1,j1)] - dpn[PTR(i2,j2)]) / 8.0 ;
		dun[PTR(i, j)] = 0.5 * uwui ;
		dvn[PTR(i, j)] = 0.0 ;
	}

	else if(i == imax && j == jmax){//‰Eã//
		int i1 = i - 1 ; int j1 = j - 1 ;
		int i2 = i - 2 ; int j2 = j - 2 ;
		dpn[PTR(i, j)] = (9.0 * dpn[PTR(i1,j1)] - dpn[PTR(i2,j2)]) / 8.0 ;
		dun[PTR(i, j)] = 0.5 * uwui ;
		dvn[PTR(i, j)] = 0.0 ;
	}
	
	else if(j == 0 && isd<=i && i<=ied){//‰º•Ó//
		int j1 = j + 1 ; int j2 = j + 2 ;
		dpn[PTR(i, j)] = (9.0 * dpn[PTR(i,j1)] - dpn[PTR(i,j2)]) / 8.0 ;
		dun[PTR(i, j)] = 0.0 ;
		dvn[PTR(i, j)] = 0.0 ;
	}

	else if(j == jmax && isd<=i && i<=ied){//ã•Ó//
		int j1 = j - 1 ; int j2 = j - 2 ;
		dpn[PTR(i, j)] = (9.0 * dpn[PTR(i,j1)] - dpn[PTR(i,j2)]) / 8.0 ;
		dun[PTR(i, j)] = uwui ;
		dvn[PTR(i, j)] = 0.0 ;
	}

	else if(i == 0 && jsd<=j && j<=jed){//¶•Ó//
		int i1 = i + 1 ; int i2 = i + 2 ;
		dpn[PTR(i, j)] = (9.0 * dpn[PTR(i1,j)] - dpn[PTR(i2,j)]) / 8.0 ;
		dun[PTR(i, j)] = 0.0 ;
		dvn[PTR(i, j)] = 0.0 ;
	}
	
	else if(i == imax && jsd<=j && j<=jed){//‰E•Ó//
		int i1 = i - 1 ; int i2 = i - 2 ;
		dpn[PTR(i, j)] = (9.0 * dpn[PTR(i1,j)] - dpn[PTR(i2,j)]) / 8.0 ;
		dun[PTR(i, j)] = 0.0 ;
		dvn[PTR(i, j)] = 0.0 ;
	}

	else{
		//’†S‹æ‰æ//
	}
	__syncthreads() ;
}


//******************************‘æ“ñ‹«ŠEˆ—‹æ‰æ********************************//
//
//
//******************************************************************************//
__global__ void CUDAboundf(PRECISION *dfe, PRECISION *dfp, PRECISION *dfn,
						   PRECISION *dpn, PRECISION *dun, PRECISION *dvn,
						   struct Conditions *d_cond)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x  ;
	const int j = blockIdx.y * blockDim.y + threadIdx.y  ;

	const int imax2 = d_cond->imax2 ;
	const int isd   = d_cond->isd ;
	const int ied   = d_cond->ied ;
	const int jsd   = d_cond->jsd ;
	const int jed   = d_cond->jed ;
	const int imax  = d_cond->imax ;
	const int jmax  = d_cond->jmax ;
	const PRECISION runi = d_cond->runi ;
	const PRECISION rcsu = d_cond->rcsu ;

	PRECISION pss , uss , vss ;
	PRECISION euv , qau ;

	if(i == 0 && j == 0){//¶‰º//
		pss = 0.0 ;
		uss = 0.0 ;
		vss = 0.0 ;
		for(int k=0;k<maa;k++){
			int ii = i + 1 ;
			int jj = j + 1 ;
			dfp[PTRQ(i,j,k)] = dfe[PTRQ(i,j,k)] + (dfn[PTRQ(ii,jj,k)] - dfe[PTRQ(ii,jj,k)]) ;
			dfp[PTRQ(i,j,k)] = 2.0 * dfp[PTRQ(i,j,k)] - dfp[PTRQ(ii,jj,k)] ;
			pss +=          dfp[PTRQ(i,j,k)] ;
			uss += dex[k] * dfp[PTRQ(i,j,k)] ;
			vss += dey[k] * dfp[PTRQ(i,j,k)] ;
		}
		dpn[PTR(i,j)] =        pss ;
		dun[PTR(i,j)] = rcsu * uss ;
		dvn[PTR(i,j)] = rcsu * vss ;
		for(int k=0;k<maa;k++){
			euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
			qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
			dfe[PTRQ(i,j,k)] = dwe[k] * (dpn[PTR(i,j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
			dfn[PTRQ(i,j,k)] = dfp[PTRQ(i,j,k)] ;
		}
	}
//
	else if(i == imax && j == 0){//‰E‰º//
		pss = 0.0 ;
		uss = 0.0 ;
		vss = 0.0 ;
		for(int k=0;k<maa;k++){
			int ii = i - 1 ;
			int jj = j + 1 ;
			dfp[PTRQ(i,j,k)] = dfe[PTRQ(i,j,k)] + (dfn[PTRQ(ii,jj,k)] - dfe[PTRQ(ii,jj,k)]) ;
			dfp[PTRQ(i,j,k)] = 2.0 * dfp[PTRQ(i,j,k)] - dfp[PTRQ(ii,jj,k)] ;
			pss +=          dfp[PTRQ(i,j,k)] ;
			uss += dex[k] * dfp[PTRQ(i,j,k)] ;
			vss += dey[k] * dfp[PTRQ(i,j,k)] ;
		}
		dpn[PTR(i,j)] =        pss ;
		dun[PTR(i,j)] = rcsu * uss ;
		dvn[PTR(i,j)] = rcsu * vss ;
		for(int k=0;k<maa;k++){
			euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
			qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
			dfe[PTRQ(i,j,k)] = dwe[k] * (dpn[PTR(i,j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
			dfn[PTRQ(i,j,k)] = dfp[PTRQ(i,j,k)] ;
		}
	}
//
	else if(i == 0 && j == jmax){//¶ã//
		pss = 0.0 ;
		uss = 0.0 ;
		vss = 0.0 ;
		for(int k=0;k<maa;k++){
			int ii = i + 1 ;
			int jj = j - 1 ;
			dfp[PTRQ(i,j,k)] = dfe[PTRQ(i,j,k)] + (dfn[PTRQ(ii,jj,k)] - dfe[PTRQ(ii,jj,k)]);
			dfp[PTRQ(i,j,k)] = 2.0 * dfp[PTRQ(i,j,k)] - dfp[PTRQ(ii,jj,k)] ;
			pss +=          dfp[PTRQ(i,j,k)] ;
			uss += dex[k] * dfp[PTRQ(i,j,k)] ;
			vss += dey[k] * dfp[PTRQ(i,j,k)] ;
		}
		dpn[PTR(i,j)] =        pss ;
		dun[PTR(i,j)] = rcsu * uss ;
		dvn[PTR(i,j)] = rcsu * vss ;
		for(int k=0;k<maa;k++){
			euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
			qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
			dfe[PTRQ(i,j,k)] = dwe[k] * (dpn[PTR(i,j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
			dfn[PTRQ(i,j,k)] = dfp[PTRQ(i,j,k)] ;
		}
	}
//
	else if(i == imax && j == jmax){//‰Eã//
		pss = 0.0 ;
		uss = 0.0 ;
		vss = 0.0 ;
		for(int k=0;k<maa;k++){
			int ii = i - 1 ;
			int jj = j - 1 ;
			dfp[PTRQ(i,j,k)] = dfe[PTRQ(i,j,k)] + (dfn[PTRQ(ii,jj,k)] - dfe[PTRQ(ii,jj,k)]) ;
			dfp[PTRQ(i,j,k)] = 2.0 * dfp[PTRQ(i,j,k)] - dfp[PTRQ(ii,jj,k)] ;
			pss +=          dfp[PTRQ(i,j,k)] ;
			uss += dex[k] * dfp[PTRQ(i,j,k)] ;
			vss += dey[k] * dfp[PTRQ(i,j,k)] ;
		}
		dpn[PTR(i,j)] =        pss ;
		dun[PTR(i,j)] = rcsu * uss ;
		dvn[PTR(i,j)] = rcsu * vss ;
		for(int k=0;k<maa;k++){
			euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
			qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
			dfe[PTRQ(i,j,k)] = dwe[k] * (dpn[PTR(i,j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
			dfn[PTRQ(i,j,k)] = dfp[PTRQ(i,j,k)] ;
		}
	}
//
	else if(j == 0 && isd<=i && i<=ied){//‰º•Ó//
		pss = 0.0 ;
		uss = 0.0 ;
		vss = 0.0 ;
		for(int k=0;k<maa;k++){
			int jj = j + 1 ;
			dfp[PTRQ(i,j,k)] = dfe[PTRQ(i,j,k)] + (dfn[PTRQ(i,jj,k)] - dfe[PTRQ(i,jj,k)]) ;
			dfp[PTRQ(i,j,k)] = 2.0 * dfp[PTRQ(i,j,k)] - dfp[PTRQ(i,jj,k)] ;
			pss +=          dfp[PTRQ(i,j,k)] ;
			uss += dex[k] * dfp[PTRQ(i,j,k)] ;
			vss += dey[k] * dfp[PTRQ(i,j,k)] ;
		}
		dpn[PTR(i,j)] =        pss ;
		dun[PTR(i,j)] = rcsu * uss ;
		dvn[PTR(i,j)] = rcsu * vss ;
		for(int k=0;k<maa;k++){
			euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
			qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
			dfe[PTRQ(i,j,k)] = dwe[k] * (dpn[PTR(i,j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
			dfn[PTRQ(i,j,k)] = dfp[PTRQ(i,j,k)] ;
		}
	}
//
	else if(j == jmax && isd<=i && i<=ied){//ã•Ó//
		pss = 0.0 ;
		uss = 0.0 ;
		vss = 0.0 ;
		for(int k=0;k<maa;k++){
			int jj = j - 1 ;
			dfp[PTRQ(i,j,k)] = dfe[PTRQ(i,j,k)] + (dfn[PTRQ(i,jj,k)] - dfe[PTRQ(i,jj,k)]) ;
			dfp[PTRQ(i,j,k)] = 2.0 * dfp[PTRQ(i,j,k)] - dfp[PTRQ(i,jj,k)] ;
			pss +=          dfp[PTRQ(i,j,k)] ;
			uss += dex[k] * dfp[PTRQ(i,j,k)] ;
			vss += dey[k] * dfp[PTRQ(i,j,k)] ;
		}
		dpn[PTR(i,j)] =        pss ;
		dun[PTR(i,j)] = rcsu * uss ;
		dvn[PTR(i,j)] = rcsu * vss ;
		for(int k=0;k<maa;k++){
			euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
			qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
			dfe[PTRQ(i,j,k)] = dwe[k] * (dpn[PTR(i,j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
			dfn[PTRQ(i,j,k)] = dfp[PTRQ(i,j,k)] ;
		}
	}
//
	else if(i == 0 && jsd<=j && j<=jed){//¶•Ó//
		pss = 0.0 ;
		uss = 0.0 ;
		vss = 0.0 ;
		for(int k=0;k<maa;k++){
			int ii = i + 1 ;
			dfp[PTRQ(i,j,k)] = dfe[PTRQ(i,j,k)] + (dfn[PTRQ(ii,j,k)] - dfe[PTRQ(ii,j,k)]) ;
			dfp[PTRQ(i,j,k)] = 2.0 * dfp[PTRQ(i,j,k)] - dfp[PTRQ(ii,j,k)] ;
			pss +=          dfp[PTRQ(i,j,k)] ;
			uss += dex[k] * dfp[PTRQ(i,j,k)] ;
			vss += dey[k] * dfp[PTRQ(i,j,k)] ;
		}
		dpn[PTR(i,j)] =        pss ;
		dun[PTR(i,j)] = rcsu * uss ;
		dvn[PTR(i,j)] = rcsu * vss ;
		for(int k=0;k<maa;k++){
			euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
			qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
			dfe[PTRQ(i,j,k)] = dwe[k] * (dpn[PTR(i,j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
			dfn[PTRQ(i,j,k)] = dfp[PTRQ(i,j,k)] ;
		}
	}
//
	else if(i == imax && jsd<=j && j<=jed){//‰E•Ó//
		pss = 0.0 ;
		uss = 0.0 ;
		vss = 0.0 ;
		for(int k=0;k<maa;k++){
			int ii = i - 1 ;
			dfp[PTRQ(i,j,k)] = dfe[PTRQ(i,j,k)] + (dfn[PTRQ(ii,j,k)] - dfe[PTRQ(ii,j,k)]) ;
			dfp[PTRQ(i,j,k)] = 2.0 * dfp[PTRQ(i,j,k)] - dfp[PTRQ(ii,j,k)] ;
			pss +=          dfp[PTRQ(i,j,k)] ;
			uss += dex[k] * dfp[PTRQ(i,j,k)] ;
			vss += dey[k] * dfp[PTRQ(i,j,k)] ;
		}
		dpn[PTR(i,j)] =        pss ;
		dun[PTR(i,j)] = rcsu * uss ;
		dvn[PTR(i,j)] = rcsu * vss ;
		for(int k=0;k<maa;k++){
			euv = dex[k] * dun[PTR(i,j)] + dey[k] * dvn[PTR(i,j)] ;
			qau = 0.5 * (dun[PTR(i,j)] * dun[PTR(i,j)] + dvn[PTR(i,j)] * dvn[PTR(i,j)]) ;
			dfe[PTRQ(i,j,k)] = dwe[k] * (dpn[PTR(i,j)] + runi * (euv + 1.5 * euv * euv - qau)) ;
			dfn[PTRQ(i,j,k)] = dfp[PTRQ(i,j,k)] ;
		}
	}
//
	else{
		//’†S‹æ‰æ//
	}
	__syncthreads() ;
}


//*****************************ƒƒCƒ“ƒvƒƒOƒ‰ƒ€*********************************//
//
//
//
//******************************************************************************//
int main()    
{

printf("This is my modification\n");

	hipGetDeviceCount(&numGPUs) ;
	if(numGPUs == 0){
	printf("No GPU detected\n") ;
		return(0) ;
	}

	hipDeviceProp_t dev;
	for (int i=0;i<numGPUs;i++){
		hipGetDeviceProperties(&dev,i) ;
		printf("Device Number :     %d\n", i);
		printf("Using device :      %s\n",       dev.name);
		printf("totalGlobalMem      %d\n",        dev.totalGlobalMem);
		printf("sharedMemPerBlock   %d\n",     dev.sharedMemPerBlock);
		printf("regsPerBlock        %d\n",          dev.regsPerBlock);
		printf("warpSize            %d\n",              dev.warpSize);
		printf("memPitch            %d\n",              dev.memPitch);
		printf("maxThreadsPerBlock  %d\n",    dev.maxThreadsPerBlock);
		printf("maxThreadsDim[3]    %d,%d,%d\n",
				dev.maxThreadsDim[0], dev.maxThreadsDim[1], dev.maxThreadsDim[2]);
		printf("maxGridSize[3]      %d,%d,%d\n",  
				dev.maxGridSize[0], dev.maxGridSize[1], dev.maxGridSize[2]);
		printf("totalConstMem       %d\n",         dev.totalConstMem);
		printf("major.minor         %d.%d\n",        dev.major, dev.minor);
		printf("clockRate           %d\n",             dev.clockRate);
		printf("textureAlignment    %d\n",      dev.textureAlignment);
		printf("deviceOverlap       %d\n",         dev.deviceOverlap);
		printf("multiProcessorCount %d\n",   dev.multiProcessorCount);
	}


	iniset();

	if(imax2%BLOCK_SIZE_X != 0 || jmax2%BLOCK_SIZE_Y != 0){
		// printf("ŠiŽq”‚ÆBLOCK_SIZE‚Æ‚ÌŠÖŒW‚ªm9(^„D^)\n") ;
		printf("Mesh number and BLOCK_SIZE is m9\n") ;
		return(0) ;
	}

	inicon();
	solver();
//	printf("‚±‚±‚ÅI‚í‚è‚¾‚æ");
	printf("Program end\n");
	return(0);
}


//******************************ŒvŽZ—p‰ŠúÝ’è**********************************//
//
//
//******************************************************************************//
void iniset()
{                              
	int i , j ;
	PRECISION rnyi , taui ;
	PRECISION w0 , w1 , w2 ;
	int xsize, ysize ;
	PRECISION chle ; //‘ã•\’·‚³//
//
	xsize = 4 ;
	ysize = 4 ;
	ima = BLOCK_SIZE_X * xsize - 2 ;
	jma = BLOCK_SIZE_Y * ysize - 2 ;
	ncye = 50000 ;
	nwri = 1000 ;
	uwui = 0.1 ;
	reyn = 100 ; //ƒŒƒCƒmƒ‹ƒY”//
//
	isd   = 1 ;
	jsd   = 1 ;
	ied   = ima ;
	jed   = jma ;
	imax  = ima + 1 ;
	jmax  = jma + 1 ;
	imax2 = imax + 1 ; 
	jmax2 = jmax + 1 ;
//
	runi = 1.0 ;
    uini = 0.0 ;
	vini = 0.0 ;
	chle = jmax ;
	rnyi = chle * uwui / reyn ;
	taui = 0.5 * (6.0 * rnyi + 1.0) ;
	rtau = 1.0 / taui ;
	csou = 1.0 / sqrt(3.0) ;
	pini = runi * csou * csou ;
	rcsu = 1.0 / pini ;

printf("Mesh size, imax=%d, jmax=%d \n", imax, jmax);

//
	ex[0] =   0.0 ;
    ex[1] =   1.0 ;
    ex[2] =   0.0 ;
	ex[3] = - 1.0 ;
    ex[4] =   0.0 ;
    ex[5] =   1.0 ;
    ex[6] = - 1.0 ;
    ex[7] = - 1.0 ;
    ex[8] =   1.0 ;
//
	ey[0] =   0.0 ;
    ey[1] =   0.0 ;
    ey[2] =   1.0 ;
    ey[3] =   0.0 ;
    ey[4] = - 1.0 ;
    ey[5] =   1.0 ;
    ey[6] =   1.0 ;
    ey[7] = - 1.0 ;
    ey[8] = - 1.0 ;
//
    w0 = 4.0 /  9.0 ;
    w1 = 1.0 /  9.0 ;
    w2 = 1.0 / 36.0 ;
    we[0] = w0 ;
    we[1] = w1 ;
    we[2] = w1 ;
    we[3] = w1 ;
    we[4] = w1 ;
    we[5] = w2 ;
    we[6] = w2 ;
    we[7] = w2 ;
    we[8] = w2 ;
//
    for(j=0;j<=jmax;j++){
		for(i=0;i<=imax;i++){
			//xg[i][j] = ((i-0.5) / ima) ;
			xg[i][j] = (i / chle) ;
			yg[i][j] = (j / chle) ;
		}
	}
}             


//*******************************‰ŠúðŒÝ’è***********************************//
//
//
//******************************************************************************//
void inicon()
{
        int i , j , k ;
        PRECISION euv , qau ;
//
        for(j=0;j<=jmax;j++){
			for(i=0;i<=imax;i++){
				pn[i][j] = pini ;
				un[i][j] = uini ;
				vn[i][j] = vini ;
			}
		}
//
        j = jmax ;
        for(i=isd;i<=ied;i++){
			pn[i][j] = pini ;
			un[i][j] = uwui ;
			vn[i][j] = vini ;
        }
//
        for(k=0;k<maa;k++){
			for(j=0;j<=jmax;j++){
				for(i=0;i<=imax;i++){
					euv = ex[k] * un[i][j] + ey[k] * vn[i][j] ;
					qau = 0.5 * ( un[i][j] * un[i][j] + vn[i][j] * vn[i][j] ) ;
					fe[i][j][k] = we[k] * (pn[i][j] + runi * (euv + 1.5*euv*euv - qau)) ;
					fn[i][j][k] = fe[i][j][k] ;
					fp[i][j][k] = 0 ;
				}
			}
        }
}

//*****************************ŽåŒvŽZƒvƒƒOƒ‰ƒ€*********************************//
//
//
//******************************************************************************//
void solver()
{
	int i, j, k, nc ;
	struct Conditions cond;
	struct Conditions *d_cond;

	PRECISION dps, duv ;
	PRECISION rda, cda, rnp, dma ;
	FILE *fop ;
//
	PRECISION matrixsize1 = sizeof(PRECISION) * maa ;
	PRECISION matrixsize2 = sizeof(PRECISION) * imax2 * jmax2 ;
	PRECISION matrixsize3 = sizeof(PRECISION) * imax2 * jmax2 * maa ;
//	cda = 1.0e-06 ;
	cda = 1.0e-05 ;
	rnp = 1.0 / ((imax - 1.0)*(jmax - 1.0)) ;
	dma = 0.0 ;

	cond.imax = imax ;
	cond.jmax = jmax ;
	cond.imax2 = imax2 ;
	cond.jmax2 = jmax2 ;
	cond.isd = isd ;
	cond.ied = ied ;
	cond.jsd = jsd ;
	cond.jed = jed ;
	cond.rtau = rtau ;
	cond.rcsu = rcsu ;
	cond.runi = runi ;
	cond.uwui = uwui ;
//
	//ƒzƒXƒg‘¤‚Ì•Ï”Ý’è//
	PRECISION *hex, *hey, *hwe, *hfn, *hfe, *hfp, *hpn, *hun, *hvn, *hdps, *hduv ;

//Marker array
PRECISION *hmarker;

//
	//ƒzƒXƒg‘¤‚Ìƒƒ‚ƒŠŠm•Û//
	hex = (PRECISION*)malloc(matrixsize1);
	hey = (PRECISION*)malloc(matrixsize1);
	hwe = (PRECISION*)malloc(matrixsize1);

	hfn = (PRECISION*)malloc(matrixsize3);
	hfe = (PRECISION*)malloc(matrixsize3);
	hfp = (PRECISION*)malloc(matrixsize3);

	hpn = (PRECISION*)malloc(matrixsize2);
	hun = (PRECISION*)malloc(matrixsize2);
	hvn = (PRECISION*)malloc(matrixsize2);

//allocating array for marker at host
	hmarker = (PRECISION*)malloc(matrixsize2);

	hdps = (PRECISION*)malloc(matrixsize2);
	hduv = (PRECISION*)malloc(matrixsize2);
//
	if (hex == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hey == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hwe == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hfn == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hfe == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hfp == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hpn == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hun == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hvn == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hdps == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}
	if (hduv == NULL) {
		printf("cannot allocate memory\n");
		return ;
	}

int ifront, iback, jdown, jtop;
ifront = 0.25*imax;
iback = 0.75*imax;
jdown = 0.25*jmax;
jtop = 0.75*jmax;

//Add initial value for hmarker
 for(i=0; i<=imax; i++){
	for(j=0; j<=jmax; j++){

if(i >= ifront && i<= iback  ){
	if(j >= jdown && j <= jtop){
	hmarker[PTR(i, j)] = 1.0;
		}
	}
else
	hmarker[PTR(i, j)] = 0.0;
	}
} 

/* FILE *fmarker;
fmarker = fopen("marker.txt","w");
for(i=0; i<= imax; i++){
	for(j=0; j<=jmax; j++){
	fprintf(fmarker, " %d %d %f\n", i, j, hmarker[PTR(i, j)]);
	}
}
fclose(fmarker); */

	//ƒzƒXƒg‚ÉŠe’l‚ð‘}“ü//
	for(j=0;j<=jmax;j++){
		for(i=0;i<=imax;i++){
			for(k=0;k<=8;k++){
				hfn[PTRQ(i, j, k)] = fn[i][j][k] ;
				hfe[PTRQ(i, j, k)] = fe[i][j][k] ;
				hfp[PTRQ(i, j, k)] = fp[i][j][k] ;
			}
		}
	}
	for(j=0;j<=jmax;j++){
		for(i=0;i<=imax;i++){
			hpn[PTR(i, j)] = pn[i][j] ;
			hun[PTR(i, j)] = un[i][j] ;
			hvn[PTR(i, j)] = vn[i][j] ;
			hdps[PTR(i, j)] = 0 ;
			hduv[PTR(i, j)] = 0 ;
		}
	}
	for(k=0;k<=8;k++){
		hex[k] = ex[k] ;
		hey[k] = ey[k] ;
		hwe[k] = we[k] ;
	}
//
	//ƒfƒoƒCƒX‘¤‚Ì•Ï”Ý’è//
	PRECISION *dfn, *dfe, *dfp, *dpn, *dun, *dvn, *ddps, *dduv ;
	PRECISION *dmarker;
//@
	//ƒfƒoƒCƒXƒƒ‚ƒŠŠm•Û‹y‚ÑƒRƒs[//
//	printf("(L¥ƒÖ¥M)\n") ; //cutilSafeCall‚Í‘‚¢‚¿‚áƒ_ƒ//
	printf("Yamazaki nuance, don't get it\n") ; //cutilSafeCall‚Í‘‚¢‚¿‚áƒ_ƒ//
	hipMemcpyToSymbol(HIP_SYMBOL(dwe), hwe, matrixsize1) ;
	hipMemcpyToSymbol(HIP_SYMBOL(dex), hex, matrixsize1) ;
	hipMemcpyToSymbol(HIP_SYMBOL(dey), hey, matrixsize1) ;
//
	hipMalloc((void**)&dfn, matrixsize3) ;
	hipMalloc((void**)&dfe, matrixsize3) ;
	hipMalloc((void**)&dfp, matrixsize3) ;

	hipMalloc((void**)&dpn, matrixsize2) ;
	hipMalloc((void**)&dun, matrixsize2) ;
	hipMalloc((void**)&dvn, matrixsize2) ;	
//allocating device memory for marker array
	hipMalloc((void**)&dmarker, matrixsize2) ;	

	hipMalloc((void**)&ddps, matrixsize2) ;
	hipMalloc((void**)&dduv, matrixsize2) ;	
	hipMalloc((void **)&d_cond, sizeof(struct Conditions)) ;
//
	hipMemcpy(dfn, hfn, matrixsize3, hipMemcpyHostToDevice) ;
	hipMemcpy(dfe, hfe, matrixsize3, hipMemcpyHostToDevice) ;
	hipMemcpy(dfp, hfp, matrixsize3, hipMemcpyHostToDevice) ;
	hipMemcpy(dpn, hpn, matrixsize2, hipMemcpyHostToDevice) ;
	hipMemcpy(dun, hun, matrixsize2, hipMemcpyHostToDevice) ;
	hipMemcpy(dvn, hvn, matrixsize2, hipMemcpyHostToDevice) ;
//copying memory from host to device for marker array
	hipMemcpy(dmarker, hmarker, matrixsize2, hipMemcpyHostToDevice) ;

	hipMemcpy(ddps, hdps, matrixsize2, hipMemcpyHostToDevice) ;
	hipMemcpy(dduv, hduv, matrixsize2, hipMemcpyHostToDevice) ;
	hipMemcpy(d_cond, &cond, sizeof(struct Conditions), hipMemcpyHostToDevice) ;
//
	//ƒuƒƒbƒNƒTƒCƒY‚ÆƒOƒŠƒbƒhƒTƒCƒY‚ÌÝ’è//
	dim3 threads(BLOCK_SIZE_X, BLOCK_SIZE_Y) ;
	dim3 block(imax2/BLOCK_SIZE_X, jmax2/BLOCK_SIZE_Y) ;
//
	//unsigned int timer = 0 ;
	//CUT_SAFE_CALL(cutCreateTimer(&timer)) ;
	//CUT_SAFE_CALL(cutStartTimer(timer)) ;
	PRECISION elapsed_time_ms = 0.0f ;
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop  );
 
	hipEventRecord( start, 0 );
//
	for(nc=1;nc<=ncye;nc++){
//original thread
//		matrix1<<<block, threads>>>(dfe, dfp, dfn, d_cond) ;
		matrix1<<<block, threads>>>(dfe, dfp, dfn, dmarker, dun, dvn, dpn, d_cond) ;
		hipDeviceSynchronize() ;

		//‰¼‘z—¬‘©–@ŒÄ‚Ño‚µˆÊ’u//

		matrix2<<<block, threads>>>(dfp, dpn, dun, dvn, ddps, dduv, d_cond) ;
		hipDeviceSynchronize() ;

		hipMemcpy(hduv, dduv, matrixsize2, hipMemcpyDeviceToHost) ;
		hipMemcpy(hdps, ddps, matrixsize2, hipMemcpyDeviceToHost) ;

		dps = 0.0 ;
		duv = 0.0 ;

		for(j=0;j<=jmax;j++){
			for(i=0;i<=imax;i++){
				dps += hdps[PTR(i, j)] ;
				duv += hduv[PTR(i, j)] ;
			}
		}

		dps = sqrt(      rnp * dps) ;
		duv = sqrt(0.5 * rnp * duv) ;

		if(duv > dma){
			dma = duv ;
		}

		rda = duv / dma ;

		CUDAboundp<<<block, threads>>>(dpn, dun, dvn, d_cond) ;
		hipDeviceSynchronize() ;

		matrix3<<<block, threads>>>(dfe, dfp, dfn, dpn, dun, dvn, d_cond) ;
		hipDeviceSynchronize() ;

		CUDAboundf<<<block, threads>>>(dfe, dfp, dfn, dpn, dun, dvn, d_cond) ;
		hipDeviceSynchronize() ;

		if(nc%nwri == 0){
	
	printf("Resids = %d %e %e %e\n",nc,dps,duv,rda);	
		}

		if(rda < cda) break ; // original definition

	}
//
	printf("Residuals at timestep= %d, L2-pressure= %e, L2-uv= %e, L2-total= %e\n",nc,dps,duv,rda) ;
	//CUT_SAFE_CALL(cutStopTimer(timer)) ;
	//printf("Processing time : %f [msec]\n", cutGetTimerValue(timer)) ;
	//CUT_SAFE_CALL(cutDeleteTimer(timer)) ;

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &elapsed_time_ms, start, stop );
 
	printf("Processing time : %f [msec]\n", elapsed_time_ms) ;

	hipEventDestroy( start );
	hipEventDestroy( stop );

	hipMemcpy(hpn, dpn, matrixsize2, hipMemcpyDeviceToHost) ;
	hipMemcpy(hun, dun, matrixsize2, hipMemcpyDeviceToHost) ;
	hipMemcpy(hvn, dvn, matrixsize2, hipMemcpyDeviceToHost) ;
	hipDeviceSynchronize() ;
	
	printf("Writing result to file\n") ;
	fop = fopen("result.plt", "w") ;
	fprintf(fop, " variables=x,y,p,u,v\n" ); 
    fprintf(fop, "zone t = flowfield\n" ); 
    fprintf(fop, "i = %d , j = %d , f=point \n"  ,ima , jma) ;
	for(j=jsd;j<=jed;j++){
		for(i=isd;i<=ied;i++){
			fprintf (fop, "%e %e %e %e %e\n",xg[i][j], yg[i][j], hpn[PTR(i, j)],
				hun[PTR(i, j)], hvn[PTR(i, j)]) ;
		}
	}
/*
    fprintf(fop, " variables=x,y,p,u,v\n " ); 
    fprintf(fop, "zone t = marker\n" ); 
    fprintf(fop, "i = %d , j = %d , f=point \n"  ,ima , jma) ;	
    for(j=jsd;j<=jed;j++){
		for(i=isd;i<=ied;i++){
			fprintf (fop, "%e %e %e %e %e\n",xg[i][j], yg[i][j], hmarker[PTR(i, j)],
				hun[PTR(i, j)], hvn[PTR(i, j)]) ;
		}
	}
*/
	fclose(fop) ;


//
	free(hex) ;
	free(hey) ;
	free(hwe) ;
	free(hfn) ;
	free(hfe) ;
	free(hfp) ;
	free(hpn) ;
	free(hun) ;
	free(hvn) ;
	free(hdps) ;
	free(hduv) ;
	
	hipFree(dfe) ;
	hipFree(dfp) ;
	hipFree(dfn) ;
	hipFree(dpn) ;
	hipFree(dun) ;
	hipFree(dvn) ;
	hipFree(ddps) ;
	hipFree(dduv) ;
}
